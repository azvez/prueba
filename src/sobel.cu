
#include <hip/hip_runtime.h>
#define tamBloque 64

__constant__ int mascaraX[3][3];
__constant__ int mascaraY[3][3];


// Kernel que se ejecutará en el grid de la GPU
__global__ void sobelKernel(float *entrada, float *salida, int filas, int columnas){
   int sumx = 0;
   int sumy = 0;
   int SUM = 0;   
   int x, y;
   int i, j, pix, piy;
   int R, G, B, NC, posR, posG, posB;
   float newPixel;
   
   y= blockIdx.y;
   x= blockIdx.x * blockDim.x + threadIdx.x;

   if (y == 0 || y == filas-1 || x==0 || x == columnas-1){
      SUM = 0;
   }
   else{
      for(i=-1; i<=1; i++)  {
         for(j=-1; j<=1; j++)  {
            pix = j + x;
            piy = i + y;

            posR= piy*columnas + pix;  // posición en el vector del componente R del pixel sobre el que trabajamos
            R = (int)entrada[posR];  // imagen(pix,piy,0,0);
            
            posG= filas*columnas + piy*columnas + pix;  // posición en el vector del componente G del pixel sobre el que trabajamos
            G = (int)entrada[posG];  // imagen(pix,piy,0,1);
            
            posB= 2*filas*columnas + piy*columnas + pix;  // posición en el vector del componente B del pixel sobre el que trabajamos
            B = (int)entrada[posB];  // imagen(pix,piy,0,2);

            NC = (R+G+B)/3;

            sumx = sumx + (NC) * mascaraX[j+1][i+1];
            sumy = sumy + (NC) * mascaraY[j+1][i+1];

         }
      }
      SUM = abs(sumx) + abs(sumy);
   }
        
  if(SUM>255){
	 SUM=255;
  }
   
   newPixel = 255 - (float)(SUM);

   salida[y*columnas + x] = newPixel;  // componente R
   salida[filas*columnas + y*columnas + x] = newPixel;  // componente G
   salida[2*filas*columnas + y*columnas + x] = newPixel;  // componente B
}



// Función que lanza la ejecución de vectores en la GPU
void calcularSobelCuda (float *hEntrada, float *hSalida, int filas, int columnas){
   
   float *dEntrada, *dSalida;
   int tam;
   dim3 DimGrid, DimBlock;

   int Gx [3][3]; int Gy [3][3];
   // Sobel Horizontal Mask
   Gx[0][0] = 1; Gx[0][1] = 0; Gx[0][2] = -1;
   Gx[1][0] = 2; Gx[1][1] = 0; Gx[1][2] = -2;
   Gx[2][0] = 1; Gx[2][1] = 0; Gx[2][2] = -1;

   // Sobel Vertical Mask
   Gy[0][0] =  1; Gy[0][1] = 2; Gy[0][2] =   1;
   Gy[1][0] =  0; Gy[1][1] = 0; Gy[1][2] =   0;
   Gy[2][0] = -1; Gy[2][1] =-2; Gy[2][2] =  -1;

   // Transferimos las máscaras a la memoria constante de la GPU
   hipMemcpyToSymbol(HIP_SYMBOL(mascaraX), Gx, 3*3*sizeof(int));
   hipMemcpyToSymbol(HIP_SYMBOL(mascaraY), Gy, 3*3*sizeof(int));
   
   // Espacio que ocupa en memoria la imagen
   tam= filas * columnas * 3 * sizeof(float);  // 3 colores (R, G, B)
   
   // Reservamos espacio y copiamos en GPU la imagen de entrada
   hipMalloc((void **) &dEntrada, tam);
   hipMemcpy(dEntrada,hEntrada,tam,hipMemcpyHostToDevice);
   
   // Reservamos espacio en GPU para la imagen de salida
   hipMalloc((void **) &dSalida, tam);
   
   // tamaño del grid y de los bloques de hebras
   DimBlock= dim3(tamBloque, 1, 1);  // bloques de tamBloque hebras
   DimGrid= dim3( ((columnas-1)/tamBloque)+1, filas, 1); // grid 2D, x= bloques necesarios para cubrir 1 fila de la imagen, y= n filas imagen
      
   // Llamada al kernel
   sobelKernel<<<DimGrid,DimBlock>>>(dEntrada,dSalida,filas,columnas);

   // Copia de resultados GPU -> host
   hipMemcpy(hSalida,dSalida,tam,hipMemcpyDeviceToHost);
   
   // Liberación de memoria en GPU
   hipFree(dEntrada);
   hipFree(dSalida);
}
   
   
   
   
